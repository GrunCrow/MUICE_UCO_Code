#include <stdio.h>
#include <hip/hip_runtime.h>

/*
                        Práctica 4

                Paola Montenegro Cantos
                Alba Márquez Rodríguez
*/

// Kernel para sumar las matrices en el device
__global__ void add(int *a, int *b, int *c, int N) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < N)
        c[index] = a[index] + b[index];
}

int main() {
    int N = 16; // número de elementos del vector
    int num_bytes = N * N * sizeof(int); // tamaño en bytes

    // punteros para el host
    int *h_a = (int*) malloc(num_bytes);
    int *h_b = (int*) malloc(num_bytes);
    int *h_c = (int*) malloc(num_bytes);

    // inicializar las matrices en el host
    for (int i = 0; i < N * N; i++) {
        h_a[i] = i;
        h_b[i] = i;
    }

    // punteros para el device
    int *d_a, *d_b, *d_c;
    hipMalloc((void**)&d_a, num_bytes);
    hipMalloc((void**)&d_b, num_bytes);
    hipMalloc((void**)&d_c, num_bytes);

    // copiar las matrices del host al device
    hipMemcpy(d_a, h_a, num_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, num_bytes, hipMemcpyHostToDevice);

    // lanzar el kernel
    add<<<N, N>>>(d_a, d_b, d_c, N * N);

    // copiar el resultado del device al host
    hipMemcpy(h_c, d_c, num_bytes, hipMemcpyDeviceToHost);

    // imprimir el resultado
    for (int i = 0; i < N * N; i++)
        printf("%d ", h_c[i]);
        printf("\n");

    // liberar la memoria en el host y en el device
    free(h_a);
    free(h_b);
    free(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
